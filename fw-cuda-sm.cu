#include<stdio.h>
#include<stdlib.h>

// CUDA Headers
#include "hip/hip_runtime.h"



// Helper definition
#define VAR(v, i) __typeof(i) v=(i)
#define FOR(i, j, k) for (int i = (j); i <= (k); ++i)
#define FORD(i, j, k)for (int i=(j); i >= (k); --i)
#define FORE(i, c) for(VAR(i, (c).begin()); i != (c).end(); ++i)
#define REP(i, n) for(int i = 0;i <(n); ++i)

// CONSTS
#define INF 	1061109567 // 3F 3F 3F 3F
#define CHARINF 63	   // 3F	
#define CHARBIT 8
#define CMCPYHTD hipMemcpyHostToDevice
#define CMCPYDTH hipMemcpyDeviceToHost

// CONSTS for compute capability 2.0
#define BLOCK_WIDTH 16
#define WARP 	    32

const bool PRINT = true; 	// print graf d or not

/** Cuda handle error, if err is not success print error and line in code
*
* @param status CUDA Error types
*/
#define HANDLE_ERROR(err) \
{ \
	if (err != hipSuccess) \
	{ \
		fprintf(stderr, "%s failed  at line %d \nError message: %s \n", \
			__FILE__, __LINE__ ,hipGetErrorString(err)); \
		exit(EXIT_FAILURE); \
	} \
}

/**Kernel for wake gpu
*
* @param reps dummy variable only to perform some action
*/
__global__ void wake_gpu_kernel(int reps) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= reps)return;
}

/**Kernel for parallel Floyd Warshall algorithm on gpu
* 
* @param u number vertex of which is performed relaxation paths [v1, v2]
* @param n number of vertices in the graph G:=(V,E), n := |V(G)|
* @param d matrix of shortest paths d(G)
*/
template <int BLOCK_SIZE> __global__ void fw_kernel(const unsigned int u, const unsigned int n, int *d)
{
	int v1 = blockDim.y * blockIdx.y + threadIdx.y;
	int v2 = blockDim.x * blockIdx.x + threadIdx.x;
	int oldValue; 
	int newValue;

	__shared__ int vu[BLOCK_SIZE]; 
	__shared__ int uv[BLOCK_SIZE];

	if (v1 < n && v2 < n)
	{
		oldValue = d[v1 * n + v2];
		if (threadIdx.y == 0) 
		{
			uv[threadIdx.x] = d[u * n + v2];
		}

		if (threadIdx.x == 0) 
		{
			vu[threadIdx.y] = d[v1 * n + u];
		}
	}

	// Synchronize to make sure the all value are loaded
	__syncthreads();

	if (v1 < n && v2 < n) 
	{
		newValue = vu[threadIdx.y] + uv[threadIdx.x];
		d[v1 * n + v2] = (oldValue  > newValue) ?  newValue : oldValue;
	}
}

/** Parallel Floyd Warshall algorithm using gpu
*
* @param n number of vertices in the graph G:=(V,E), n := |V(G)|
* @param G is a the graph G:=(V,E)
* @param d matrix of shortest paths d(G)
*/
hipError_t fw_gpu(const unsigned int n, int *G, int *d)
{
	int *dev_d = 0;
	hipError_t cudaStatus;
	hipStream_t cpyStream;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	HANDLE_ERROR(cudaStatus);

	// Initialize the grid and block dimensions here
	dim3 dimGrid((n - 1) / BLOCK_WIDTH + 1, (n - 1) / BLOCK_WIDTH + 1, 1); 
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);

	#ifdef DEBUG
		printf("|V| %d\n", n);
		printf("Dim Grid:\nx - %d\ny -%d\nz - %d\n", dimGrid.x, dimGrid.y, dimGrid.z);
		printf("Dim Block::\nx - %d\ny -%d\nz - %d\n", dimBlock.x, dimBlock.y, dimBlock.z);
	#endif
	
	// Wake up gpu 
 	wake_gpu_kernel<<<1, dimBlock>>>(32);
  
	// Create new stream to copy data	
	cudaStatus = hipStreamCreate(&cpyStream);
	HANDLE_ERROR(cudaStatus);

	// Allocate GPU buffers for matrix of shortest paths d(G)
	cudaStatus =  hipMalloc((void**)&dev_d, n * n * sizeof(int));
	HANDLE_ERROR(cudaStatus);

        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpyAsync(dev_d, G, n * n * sizeof(int), CMCPYHTD, cpyStream);
        HANDLE_ERROR(cudaStatus);

        // cudaDeviceSynchronize waits for the kernel to finish, and returns
        cudaStatus = hipDeviceSynchronize();
        HANDLE_ERROR(cudaStatus);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(fw_kernel<BLOCK_WIDTH>), hipFuncCachePreferL1);
	FOR(u, 0, n - 1) 
	{
		fw_kernel<BLOCK_WIDTH><<<dimGrid, dimBlock>>>(u, n, dev_d);
	}

	// Check for any errors launching the kernel
    	cudaStatus = hipGetLastError();
	HANDLE_ERROR(cudaStatus);

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	HANDLE_ERROR(cudaStatus);

	cudaStatus = hipMemcpy(d, dev_d, n * n * sizeof(int), CMCPYDTH);
	HANDLE_ERROR(cudaStatus);

	cudaStatus = hipFree(dev_d);
	return cudaStatus;
}

/**
* Print graf G as a matrix
*
* @param n number of vertices in the graph G:=(V,E), n := |V(G)|
* @param G is a the graph G:=(V,E)
*/
void print_graf(const unsigned int n, const int *G)
{
	FOR(v1, 0, n - 1)
	{
		FOR(v2, 0, n - 1) 
		{	
			if (G[v1 * n + v2] < INF)
				printf("%d ", G[v1 * n + v2]);
			else
				printf("INF ");
		}
		printf("\n");
	}
}

int main(int argc, char **argv)
{
	unsigned int V;
	unsigned int E;
	unsigned int v1, v2, w; 
	
	// Load number vertices of the graph |V(G)| and number edges of the graph |E(G)|
	scanf("%d %d", &V, &E);
		
	// Alloc host data for G - graf, d - matrix of shortest paths
	unsigned int size = V * V;
	
	int *G = (int *) malloc (sizeof(int) * size);
	int *d = (int *) malloc (sizeof(int) * size);
	
	// Init Data for the graf G
	memset(G, CHARINF, sizeof(int) * V * V);
	
	#ifdef DEBUG
		print_graf(V, G);
	#endif

	// Load weight of the edges of the graph E(G)
	REP(e, E)
	{
		scanf("%d %d %d", &v1, &v2, &w);
		G[v1 * V + v2] = w;
	}

	FOR (v, 0, V - 1)
		G[v * V + v] = 0;

	#ifdef DEBUG
		print_graf(V, G);
	#endif

  	fw_gpu(V, G, d);

	if (PRINT) print_graf(V, d);
 
	// Delete allocated memory 
	free(G);
	free(d);

	return 0;
}
